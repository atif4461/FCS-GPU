/*
  Copyright (C) 2002-2021 CERN for the benefit of the ATLAS collaboration
*/

#include "LoadGpuFuncHist.h"
#include <iostream>
#include "gpuQ.h"
#include "DEV_BigMem.h"

DEV_BigMem* DEV_BigMem::bm_ptr;

LoadGpuFuncHist::LoadGpuFuncHist() {}

LoadGpuFuncHist::~LoadGpuFuncHist() {
  free( m_hf );
  hipFree( m_hf_d );

  free( m_hf2d );
  free( m_hf2d_d );
}

/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */

void LoadGpuFuncHist::LD2D() {
  if ( !m_hf2d ) {
    std::cout << "Error Load 2DFunctionHisto " << std::endl;
    return;
  }

  FH2D* hf_ptr = new FH2D;
  FH2D  hf     = {0, 0, 0, 0, 0};

  hf.nbinsx = ( *m_hf2d ).nbinsx;
  hf.nbinsy = ( *m_hf2d ).nbinsy;

  DEV_BigMem* p = DEV_BigMem::bm_ptr;

  hf.h_bordersx = (float*)( p->dev_bm_alloc( ( hf.nbinsx + 1 ) * sizeof( float ) ) );
  hf.h_bordersy = (float*)( p->dev_bm_alloc( ( hf.nbinsy + 1 ) * sizeof( float ) ) );
  hf.h_contents = (float*)( p->dev_bm_alloc( hf.nbinsy * hf.nbinsx * sizeof( float ) ) );

  gpuQ( hipMemcpy( hf.h_bordersx, ( *m_hf2d ).h_bordersx, ( hf.nbinsx + 1 ) * sizeof( float ),
                    hipMemcpyHostToDevice ) );
  gpuQ( hipMemcpy( hf.h_bordersy, ( *m_hf2d ).h_bordersy, ( hf.nbinsy + 1 ) * sizeof( float ),
                    hipMemcpyHostToDevice ) );
  gpuQ( hipMemcpy( hf.h_contents, ( *m_hf2d ).h_contents, ( hf.nbinsx * hf.nbinsy ) * sizeof( float ),
                    hipMemcpyHostToDevice ) );

  *( hf_ptr ) = hf;
  m_hf2d_d    = hf_ptr;

  m_d_hf2d = (FH2D*)( p->dev_bm_alloc( sizeof( FH2D ) ) );
  gpuQ( hipMemcpy( m_d_hf2d, m_hf2d_d, sizeof( FH2D ), hipMemcpyHostToDevice ) );
}

/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */

void LoadGpuFuncHist::LD() {
  // this call  assume  already have Histofuncs set in m_hf
  // this function allocate memory of GPU and deep copy m_hf to m_hf_d

  if ( !m_hf ) {
    std::cout << "Error Load WiggleHistoFunctions " << std::endl;
    return;
  }

  FHs hf              = {0, 0, 0, 0, 0, 0};
  hf.s_MaxValue       = ( *m_hf ).s_MaxValue;
  hf.nhist            = ( *m_hf ).nhist;
  unsigned int* h_szs = ( *m_hf ).h_szs; // already allocateded on host ;

  DEV_BigMem* p = DEV_BigMem::bm_ptr;

  hf.low_edge = (float*)( p->dev_bm_alloc( ( hf.nhist + 1 ) * sizeof( float ) ) );
  gpuQ( hipMemcpy( hf.low_edge, ( *m_hf ).low_edge, ( hf.nhist + 1 ) * sizeof( float ), hipMemcpyHostToDevice ) );

  hf.h_szs = (unsigned int*)( p->dev_bm_alloc( hf.nhist * sizeof( float ) ) );
  gpuQ( hipMemcpy( hf.h_szs, ( *m_hf ).h_szs, hf.nhist * sizeof( unsigned int ), hipMemcpyHostToDevice ) );

  hf.h_contents = (uint32_t**)( p->dev_bm_alloc( hf.nhist * sizeof( uint32_t* ) ) );
  hf.h_borders  = (float**)( p->dev_bm_alloc( hf.nhist * sizeof( float* ) ) );

  uint32_t** contents_ptr = (uint32_t**)malloc( hf.nhist * sizeof( uint32_t* ) );
  float**    borders_ptr  = (float**)malloc( hf.nhist * sizeof( float* ) );

  for ( unsigned int i = 0; i < hf.nhist; ++i ) {

    contents_ptr[i] = (uint32_t*)( p->dev_bm_alloc( h_szs[i] * sizeof( uint32_t ) ) );
    borders_ptr[i]  = (float*)( p->dev_bm_alloc( ( h_szs[i] + 1 ) * sizeof( float ) ) );

    gpuQ(
        hipMemcpy( contents_ptr[i], ( *m_hf ).h_contents[i], h_szs[i] * sizeof( uint32_t ), hipMemcpyHostToDevice ) );
    gpuQ( hipMemcpy( borders_ptr[i], ( *m_hf ).h_borders[i], ( h_szs[i] + 1 ) * sizeof( float ),
                      hipMemcpyHostToDevice ) );

    // for (int j=0; j<h_szs[i]; ++j) {
    //   std::cout << i << " " << j << " " << m_hf->h_contents[i][j] << " " << m_hf->h_borders[i][j]
    //             << " hbrds\n";
    // }
    
  }

  gpuQ( hipMemcpy( hf.h_contents, contents_ptr, hf.nhist * sizeof( uint32_t* ), hipMemcpyHostToDevice ) );
  gpuQ( hipMemcpy( hf.h_borders, borders_ptr, hf.nhist * sizeof( float* ), hipMemcpyHostToDevice ) );

  m_d_hf = (FHs*)( p->dev_bm_alloc( sizeof( FHs ) ) );
  gpuQ( hipMemcpy( m_d_hf, &hf, sizeof( FHs ), hipMemcpyHostToDevice ) );

  free( contents_ptr );
  free( borders_ptr );

  m_hf_d = &hf;

  // std::cout << "LD1D: nhist: "<<hf.nhist<<"   memeory: " <<s << " M of FHs str: "<< sizeof(FHs)  <<std::endl ;

  // std::cout << "================== LoadGpuFuncHist::LD =======================\n";
  // std::cout << m_hf_h->nhist << "\n";
  // for (int i=0; i<m_hf_h->nhist; ++i) {
  //   std::cout << "h_szs: " << m_hf_h->h_szs[i] << "\n";
  //   for (int j=0; j<m_hf_h->mxsz; ++j) {
  //     std::cout << "  " << m_hf_h->h_contents[i][j] << "\n";
  //   }
  // }
}
