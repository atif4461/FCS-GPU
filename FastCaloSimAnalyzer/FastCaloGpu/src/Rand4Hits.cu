#include "Rand4Hits.h"

float *  Rand4Hits::HitsRandGen(unsigned int nhits, unsigned long long seed ) {

  gpuQ(hipMalloc((void**)&m_rand_ptr , 3*nhits*sizeof(float))) ;
  CURAND_CALL(hiprandCreateGenerator(&m_gen, 
                HIPRAND_RNG_PSEUDO_DEFAULT));
  CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(m_gen, seed)) ;

  CURAND_CALL(hiprandGenerateUniform(m_gen, m_rand_ptr, 3*nhits));

   return m_rand_ptr ;
} 


